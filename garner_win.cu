
#include <hip/hip_runtime.h>
#define ulong unsigned long long
#define uint unsigned int

#define MOD_P0 469762049LL
#define MOD_P1 1811939329LL
#define MOD_P2 2013265921LL


//３つの互いに素なPが与えられるので、それぞれの余りから元の値を復元したい
//このときPは全て固定なので剰余計算は全部決め打ちでいける

//E0〜E2が入力、E3に出力
//繰り上がりも考慮
//arrayLength2=arrayE3の配列サイズ
__global__ void GarnerGPU(uint *arrayE0,uint *arrayE1,uint *arrayE2,uint *arrayE3,uint arrayLength2 ) {
	int idx = threadIdx.x+blockIdx.x*256;
	
	ulong ar=arrayE0[idx];
	ulong br=arrayE1[idx];
	ulong cr=arrayE2[idx];
	
	ulong x=ar;
	ulong brx=br-x+MOD_P1;
	if (brx>=MOD_P1)brx-=MOD_P1;
	x=x+(brx*1540148431)%MOD_P1*MOD_P0;
	//1540148431=modinv(MOD_P0,MOD_P1)
	//この時点でxはMOD_P1*MOD_P0以下であることが保証されている

	ulong crx=cr+MOD_P2-x%MOD_P2;
	if (crx>=MOD_P2)crx-=MOD_P2;
	ulong w1=(crx*1050399624)%MOD_P2;
	//1050399624=modinv(MOD_P0,MOD_P2) *modinv(MOD_P1,MOD_P2)%MOD_P2
	ulong w2=MOD_P0*MOD_P1;
	ulong out_lo=w1*w2;
	ulong out_hi=__umul64hi(w1,w2);
	
	if (out_lo>(out_lo+x)){
		out_hi++;
	}
	out_lo+=x;
	
	//ここから繰り上がり処理
	uint ui00_32=(uint)(out_lo%(1ULL<<32ULL));
	uint ui32_64=(uint)(out_lo/(1ULL<<32ULL));
	uint ui64_96=(uint)(out_hi%(1ULL<<32ULL));
	
	uint lastE3_0 = atomicAdd( &arrayE3[idx+0], ui00_32 );
	if ((lastE3_0+ui00_32)<lastE3_0){//繰り上がりを考慮
		ui32_64++;
		if (ui32_64==0)ui64_96++;
	}
	
	if (ui32_64!=0){
		uint lastE3_1 = atomicAdd( &arrayE3[idx+1], ui32_64 );
		if ((lastE3_1+ui32_64)<lastE3_1){//繰り上がりを考慮
			ui64_96++;//こいつがオーバーフローすることは絶対にない
		}
	}
	
	uint upflg=0;
	if (ui64_96!=0){
		uint lastE3_2 = atomicAdd( &arrayE3[idx+2], ui64_96 );
		if ((lastE3_2+ui64_96)<lastE3_2){//繰り上がりを考慮
			upflg++;
		}
	}
	
	uint lastE3_i;
	for(int i=idx+3;i<arrayLength2;i++){ //9999999+1みたいなとき用
		if (upflg==0)break;
		lastE3_i = atomicAdd( &arrayE3[i], upflg );
		if (lastE3_i==4294967295){
			upflg=1;
		}else{
			upflg=0;
		}
	}
}